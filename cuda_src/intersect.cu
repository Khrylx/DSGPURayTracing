#include "hip/hip_runtime.h"
__device__ inline bool bboxIntersect(const GPUBBox *bbox, GPURay& r, float& t0, float& t1) {
    for (int i = 0; i < 3; ++i) {
        if (r.d[i] != 0.0) {
            double tx1 = (bbox->min[i] - r.o[i]) / r.d[i];
            double tx2 = (bbox->max[i] - r.o[i]) / r.d[i];

            t0 = fmaxf(t0, fminf(tx1, tx2));
            t1 = fminf(t1, fmaxf(tx1, tx2));
        }
    }

    return t0 <= t1;
}

// primitive and normals are shift pointers to the primitive and normal we selected
__device__ inline bool triangleIntersect(int primIndex, GPURay& r) {

    float* primitive = const_params.positions + 9 * primIndex;

    float* v1 = primitive;
    float* v2 = primitive + 3;
    float* v3 = primitive + 6;

    float e1[3], e2[3], s[3];
    subVector3D(v2, v1, e1);
    subVector3D(v3, v1, e2);
    subVector3D(r.o, v1, s);

    float tmp[3];
    VectorCross3D(e1, r.d, tmp);
    double f = VectorDot3D(tmp, e2);
    if (f == 0) {
        return false;
    }

    VectorCross3D(s, r.d, tmp);
    double u = VectorDot3D(tmp, e2) / f;
    VectorCross3D(e1, r.d, tmp);
    double v = VectorDot3D(tmp, s) / f;
    VectorCross3D(e1, s, tmp);
    double t = - VectorDot3D(tmp, e2) / f;

    if (u >= 0 && v >= 0 && u+v <= 1 && t > r.min_t && t < r.max_t) {
        return true;
    }

    return false;
}

// primitive and normals are shift pointers to the primitive and normal we selected
__device__ inline bool triangleIntersect(int primIndex, GPURay& r, GPUIntersection *isect) {

    float* primitive = const_params.positions + 9 * primIndex;
    float* normals = const_params.normals + 9 * primIndex;

    float* v1 = primitive;
    float* v2 = primitive + 3;
    float* v3 = primitive + 6;

    float e1[3], e2[3], s[3];
    subVector3D(v2, v1, e1);
    subVector3D(v3, v1, e2);
    subVector3D(r.o, v1, s);

    float tmp[3];
    VectorCross3D(e1, r.d, tmp);
    double f = VectorDot3D(tmp, e2);
    if (f == 0) {
        return false;
    }

    VectorCross3D(s, r.d, tmp);
    double u = VectorDot3D(tmp, e2) / f;
    VectorCross3D(e1, r.d, tmp);
    double v = VectorDot3D(tmp, s) / f;
    VectorCross3D(e1, s, tmp);
    double t = - VectorDot3D(tmp, e2) / f;

    if (!(u >= 0 && v >= 0 && u+v <= 1 && t > r.min_t && t < r.max_t && t < isect->t)) {
        return false;
    }

    r.max_t = t;

    isect->bsdfIndex = const_params.bsdfIndexes[primIndex];
    isect->t = t;
    isect->pIndex = primIndex;

    float *n1 = normals;
    float *n2 = normals + 3;
    float *n3 = normals + 6;

    float n[3];
    for (int i = 0; i < 3; ++i)
    {
        n[i] = (1 - u - v) * n1[i] + u * n2[i] + v * n3[i];
    }
    if (VectorDot3D(r.d, n) > 0)
    {
        negVector3D(n, n);
    }
    readVector3D(n, isect->n);

    return true;
}

__device__ inline bool sphereTest(int primIndex, GPURay& ray, double& t1, double& t2) {
    float* primitive = const_params.positions + 9 * primIndex;
    float* o = primitive;
    float r = primitive[3];
    float r2 = r * r;

    float m[3];
    subVector3D(o, ray.o, m);
    double b = VectorDot3D(m, ray.d);
    double c = VectorDot3D(m, m) - r2;
    double delta = b * b - c;
    if (delta < 0) {
        return false;
    }

    t1 = b - sqrt(delta);
    t2 = b + sqrt(delta);

    if (t1 >= ray.max_t || t2 <= ray.min_t) {
        return false;
    }

    return true;
}

__device__ inline bool sphereIntersect(int primIndex, GPURay& r) {
    double tmp;
    return sphereTest(primIndex, r, tmp, tmp);
}

__device__ inline bool sphereIntersect(int primIndex, GPURay& r, GPUIntersection *isect) {
    double t1;
    double t2;
    bool res = sphereTest(primIndex, r, t1, t2);
    if (!res) {
        return false;
    }
    isect->bsdfIndex = const_params.bsdfIndexes[primIndex];
    isect->pIndex = primIndex;

    float* primitive = const_params.positions + 9 * primIndex;
    float* o = primitive;
    double t = t1;
    if (t1 <= r.min_t) {
        t = t2;
    }
    float n[3];
    float tmp[3];
    for (int i = 0; i < 3; ++i)
    {
        tmp[i] = r.d[i] * t;
    }
    addVector3D(r.o, tmp);
    subVector3D(tmp, o, n);
    normalize3D(n);
    readVector3D(n, isect->n);
    isect->t = t;
    r.max_t = t;

    return true;
}

__device__ inline bool intersect(int primIndex, GPURay& r) {
    if (const_params.types[primIndex] == 0) {
        // sphere
        return sphereIntersect(primIndex, r);
    } else {
        // triangle
        return triangleIntersect(primIndex, r);
    }
}

__device__ inline bool intersect(int primIndex, GPURay& r, GPUIntersection *isect) {
    if (const_params.types[primIndex] == 0) {
        // sphere
        return sphereIntersect(primIndex, r, isect);
    } else {
        // triangle
        return triangleIntersect(primIndex, r, isect);
    }
}

// intersect with BVHNode
__device__ bool node_intersect(const GPUBVHNode *node, GPURay &ray, GPUIntersection *i) {
    if (node->left == NULL && node->right == NULL) {
        bool isIntersect = false;
        for (int j = 0; j < node->range; j++) {
            int primIndex = const_params.BVHPrimMap[node->start + j];
            bool res = intersect(primIndex, ray, i);
            // bool res = false;
            isIntersect = isIntersect || res;
        }
        return isIntersect;
    } else if (node->left == NULL) {
        return node_intersect(node->right, ray, i);
    } else if (node->right == NULL) {
        return node_intersect(node->left, ray, i);
    } else {
        float tminl = -INF_FLOAT;
        float tminr = -INF_FLOAT;
        float tmaxl = INF_FLOAT;
        float tmaxr = INF_FLOAT;

        GPURay nray = ray;
        float eps[3] = {EPS_K, EPS_K, EPS_K};
        addVector3D(eps, nray.d);
        normalize3D(nray.d);

        bool hitl = bboxIntersect(&(node->left->bbox), nray, tminl, tmaxl);
        bool hitr = bboxIntersect(&(node->right->bbox), nray, tminr, tmaxr);

        if (hitl && hitr) {
            GPUBVHNode* first = (tminl <= tminr) ? node->left : node->right;
            GPUBVHNode* second = (tminl <= tminr) ? node->right : node->left;

            hitl = node_intersect(first, ray, i);
            if (!hitl || i->t > fmaxf(tminl, tminr)) {
                hitr = node_intersect(second, ray, i);
            }
            return hitl || hitr;
        } else if (hitl) {
            return node_intersect(node->left, ray, i);
        } else if (hitr) {
            return node_intersect(node->right, ray, i);
        }

        return false;

    }

}

__device__ bool node_intersect_iter(const GPUBVHNode *node, GPURay &ray, GPUIntersection *i) {
    
    GPUBVHNode* stack[64];
    GPUBVHNode** stackPtr = stack;
    *stackPtr++ = NULL;

    bool isIntersect = false;

    while(1){

        while(node){
            if (node->left == NULL && node->right == NULL) {
                break;
            }
            else {
                float tminl = -INF_FLOAT;
                float tminr = -INF_FLOAT;
                float tmaxl = INF_FLOAT;
                float tmaxr = INF_FLOAT;

                GPURay nray = ray;
                float eps[3] = {EPS_K, EPS_K, EPS_K};
                addVector3D(eps, nray.d);
                normalize3D(nray.d);

                bool hitl = bboxIntersect(&(node->left->bbox), nray, tminl, tmaxl);
                bool hitr = bboxIntersect(&(node->right->bbox), nray, tminr, tmaxr);

                if (hitl && hitr) {
                    GPUBVHNode* first = (tminl <= tminr) ? node->left : node->right;
                    GPUBVHNode* second = (tminl <= tminr) ? node->right : node->left;

                    node = first;
                    *stackPtr++ = second;
                } else if (hitl) {
                    node = node->left;
                } else if (hitr) {
                    node = node->right;
                }
                else{
                    node = *--stackPtr;
                }
            }
        }
        
        if (node == NULL) {
            return isIntersect;
        }

        for (int j = 0; j < node->range; j++) {
            int primIndex = const_params.BVHPrimMap[node->start + j];
            bool res = intersect(primIndex, ray, i);
            // bool res = false;
            isIntersect = isIntersect || res;
        }
        node = *--stackPtr;
    }
    

}


__device__ bool node_intersect(GPUBVHNode *node, GPURay &ray) {
    if (node == NULL) {
        return false;
    }
    float t0 = -INF_FLOAT;
    float t1 = INF_FLOAT;

    if (!bboxIntersect(&(node->bbox), ray, t0, t1)) {
        return false;
    }

    if (node->left == NULL && node->right == NULL) {
        // node is leaf
        for (int i = 0; i < node->range; i++) {
            int primIndex = const_params.BVHPrimMap[node->start + i];
            if (intersect(primIndex, ray)) {
                return true;
            }
        }
        return false;
    } else {
        return node_intersect(node->left, ray) || node_intersect(node->right, ray);
    }
}

__device__ bool node_intersect_iter(GPUBVHNode *node, GPURay &ray) {
    GPUBVHNode* stack[64];
    GPUBVHNode** stackPtr = stack;
    *stackPtr++ = NULL;


    while(1){

        while(node){
            if (node->left == NULL && node->right == NULL) {
                break;
            }
            else {
                float tminl = -INF_FLOAT;
                float tminr = -INF_FLOAT;
                float tmaxl = INF_FLOAT;
                float tmaxr = INF_FLOAT;

                GPURay nray = ray;
                float eps[3] = {EPS_K, EPS_K, EPS_K};
                addVector3D(eps, nray.d);
                normalize3D(nray.d);

                bool hitl = bboxIntersect(&(node->left->bbox), nray, tminl, tmaxl);
                bool hitr = bboxIntersect(&(node->right->bbox), nray, tminr, tmaxr);

                if (hitl && hitr) {
                    GPUBVHNode* first = (tminl <= tminr) ? node->left : node->right;
                    GPUBVHNode* second = (tminl <= tminr) ? node->right : node->left;

                    node = first;
                    *stackPtr++ = second;
                } else if (hitl) {
                    node = node->left;
                } else if (hitr) {
                    node = node->right;
                }
                else{
                    node = *--stackPtr;
                }
            }
        }
        
        if (node == NULL) {
            return false;
        }

        for (int j = 0; j < node->range; j++) {
            int primIndex = const_params.BVHPrimMap[node->start + j];
            if(intersect(primIndex, ray))
                return true;
        }
        node = *--stackPtr;
    }
}

__device__ inline bool BVH_intersect(GPURay &ray, GPUIntersection *isect) {
    return node_intersect_iter(const_params.BVHRoot, ray, isect);
}

__device__ inline bool BVH_intersect(GPURay &ray) {
    return node_intersect_iter(const_params.BVHRoot, ray);
}
