#include "hip/hip_runtime.h"
__device__ inline bool bboxIntersect(GPUBBox *bbox, GPURay& r, double& t0, double& t1) {
    for (int i = 0; i < 3; ++i) {
        if (r.d[i] != 0.0) {
            double tx1 = (bbox->min[i] - r.o[i]) / r.d[i];
            double tx2 = (bbox->max[i] - r.o[i]) / r.d[i];

            t0 = fmaxf(t0, fminf(tx1, tx2));
            t1 = fminf(t1, fmaxf(tx1, tx2));
        }
    }

    return t0 <= t1;
}

// primitive and normals are shift pointers to the primitive and normal we selected
__device__ inline bool triangleIntersect(int primIndex, GPURay& r) {

    float* primitive = const_params.positions + 9 * primIndex;

    float* v1 = primitive;
    float* v2 = primitive + 3;
    float* v3 = primitive + 6;

    float e1[3], e2[3], s[3];
    subVector3D(v2, v1, e1);
    subVector3D(v3, v1, e2);
    subVector3D(r.o, v1, s);

    float tmp[3];
    VectorCross3D(e1, r.d, tmp);
    double f = VectorDot3D(tmp, e2);
    if (f == 0) {
        return false;
    }

    VectorCross3D(s, r.d, tmp);
    double u = VectorDot3D(tmp, e2) / f;
    VectorCross3D(e1, r.d, tmp);
    double v = VectorDot3D(tmp, s) / f;
    VectorCross3D(e1, s, tmp);
    double t = - VectorDot3D(tmp, e2) / f;

    if (u >= 0 && v >= 0 && u+v <= 1 && t > r.min_t && t < r.max_t) {
        return true;
    }

    return false;
}

// primitive and normals are shift pointers to the primitive and normal we selected
__device__ inline bool triangleIntersect(int primIndex, GPURay& r, GPUIntersection *isect) {

    float* primitive = const_params.positions + 9 * primIndex;
    float* normals = const_params.normals + 9 * primIndex;

    float* v1 = primitive;
    float* v2 = primitive + 3;
    float* v3 = primitive + 6;

    float e1[3], e2[3], s[3];
    subVector3D(v2, v1, e1);
    subVector3D(v3, v1, e2);
    subVector3D(r.o, v1, s);

    float tmp[3];
    VectorCross3D(e1, r.d, tmp);
    double f = VectorDot3D(tmp, e2);
    if (f == 0) {
        return false;
    }

    VectorCross3D(s, r.d, tmp);
    double u = VectorDot3D(tmp, e2) / f;
    VectorCross3D(e1, r.d, tmp);
    double v = VectorDot3D(tmp, s) / f;
    VectorCross3D(e1, s, tmp);
    double t = - VectorDot3D(tmp, e2) / f;

    if (!(u >= 0 && v >= 0 && u+v <= 1 && t > r.min_t && t < r.max_t && t < isect->t)) {
        return false;
    }

    r.max_t = t;

    isect->bsdfIndex = const_params.bsdfIndexes[primIndex];
    isect->t = t;
    isect->pIndex = primIndex;

    float *n1 = normals;
    float *n2 = normals + 3;
    float *n3 = normals + 6;

    float n[3];
    for (int i = 0; i < 3; ++i)
    {
        n[i] = (1 - u - v) * n1[i] + u * n2[i] + v * n3[i];
    }
    if (VectorDot3D(r.d, n) > 0)
    {
        negVector3D(n, n);
    }
    readVector3D(n, isect->n);

    return true;
}

__device__ inline bool sphereTest(int primIndex, GPURay& ray, double& t1, double& t2) {
    float* primitive = const_params.positions + 9 * primIndex;
    float* o = primitive;
    float r = primitive[3];
    float r2 = r * r;

    float m[3];
    subVector3D(o, ray.o, m);
    double b = VectorDot3D(m, ray.d);
    double c = VectorDot3D(m, m) - r2;
    double delta = b * b - c;
    if (delta < 0) {
        return false;
    }

    t1 = b - sqrt(delta);
    t2 = b + sqrt(delta);

    if (t1 >= ray.max_t || t2 <= ray.min_t) {
        return false;
    }

    return true;
}

__device__ inline bool sphereIntersect(int primIndex, GPURay& r) {
    double tmp;
    return sphereTest(primIndex, r, tmp, tmp);
}

__device__ inline bool sphereIntersect(int primIndex, GPURay& r, GPUIntersection *isect) {
    double t1;
    double t2;
    bool res = sphereTest(primIndex, r, t1, t2);
    if (!res) {
        return false;
    }
    isect->bsdfIndex = const_params.bsdfIndexes[primIndex];
    isect->pIndex = primIndex;

    float* primitive = const_params.positions + 9 * primIndex;
    float* o = primitive;
    double t = t1;
    if (t1 <= r.min_t) {
        t = t2;
    }
    float n[3];
    float tmp[3];
    for (int i = 0; i < 3; ++i)
    {
        tmp[i] = r.d[i] * t;
    }
    addVector3D(r.o, tmp);
    subVector3D(tmp, o, n);
    normalize3D(n);
    readVector3D(n, isect->n);
    isect->t = t;
    r.max_t = t;

    return true;
}

__device__ inline bool intersect(int primIndex, GPURay& r) {
    if (const_params.types[primIndex] == 0) {
        // sphere
        return sphereIntersect(primIndex, r);
    } else {
        // triangle
        return triangleIntersect(primIndex, r);
    }
}

__device__ inline bool intersect(int primIndex, GPURay& r, GPUIntersection *isect) {
    if (const_params.types[primIndex] == 0) {
        // sphere
        return sphereIntersect(primIndex, r, isect);
    } else {
        // triangle
        return triangleIntersect(primIndex, r, isect);
    }
}
