#include "hip/hip_runtime.h"
#include <stdio.h>

#include "helper.cu"
#include "setup.h"

#define MAX_NUM_LIGHT 20
#define MAX_NUM_BSDF 20

__constant__  GPUCamera const_camera;
__constant__  GPUBSDF const_bsdfs[MAX_NUM_BSDF];
__constant__  GPULight const_lights[MAX_NUM_LIGHT];
__constant__  Parameters const_params;


__device__ void
generateRay(GPURay* ray, float x, float y)
{
    float sp[3];
    sp[0] = -(x-0.5) * const_camera.widthDivDist;
    sp[1] = -(y-0.5) * const_camera.heightDivDist;
    sp[2] = 1;
    float dir[3];
    dir[0] = -sp[0];
    dir[1] = -sp[1];
    dir[2] = -sp[2];
    float world_sp[3];
    MatrixMulVector3D(const_camera.c2w, sp, world_sp);
    
}

__device__ float3
tracePixel(int x, int y)
{
    float3 s;
    
    int w = const_params.screenW;
    int h = const_params.screenH;
    
    float px = x / (float)w;
    float py = y / (float)h;
    
    GPURay ray;
    generateRay(&ray, px, py);
}


__global__ void
traceScene()
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index >= const_params.screenW * const_params.screenH) {
        return;
    }
    
    int x = index % const_params.screenW;
    int y = index / const_params.screenW;
    
    tracePixel(x, y);
    
    const_params.frameBuffer[3 * index] = 1.0;
    const_params.frameBuffer[3 * index + 1] = 0.5;
    const_params.frameBuffer[3 * index + 2] = 0.5;
}





__global__ void
vectorAdd(float *A, float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        gpuAdd(A + i, B + i, C + i);
        //C[i] = A[i] + B[i];
    }
}

__global__ void
printInfo()
{
    GPUBSDF* bsdfs = const_bsdfs;
    GPUCamera camera = const_camera;
    
    for (int i = 0; i < 8; i++) {
        if (bsdfs[i].type == 0) {
            printf("0: %lf %lf %lf\n", bsdfs[i].albedo[0], bsdfs[i].albedo[1], bsdfs[i].albedo[2] );
        }
        else if (bsdfs[i].type == 1) {
            printf("1: %lf %lf %lf\n", bsdfs[i].reflectance[0], bsdfs[i].reflectance[1], bsdfs[i].reflectance[2] );
        }
        else if (bsdfs[i].type == 2) {
            //cout << "2" << endl;
        }
        else if (bsdfs[i].type == 3) {
            printf("3: %lf %lf %lf\n", bsdfs[i].reflectance[0], bsdfs[i].reflectance[1], bsdfs[i].reflectance[2] );
            printf("3: %lf %lf %lf\n", bsdfs[i].transmittance[0], bsdfs[i].transmittance[1], bsdfs[i].transmittance[2] );
        }
        else {
            printf("4: %lf %lf %lf\n", bsdfs[i].albedo[0], bsdfs[i].albedo[1], bsdfs[i].albedo[2] );
        }
    }
    
    
    printf("%lf %lf %lf\n", camera.pos[0], camera.pos[1], camera.pos[2] );
    
    
    float* positions = const_params.positions;
    float* normals = const_params.normals;
    
    printf("+++++++++++++++++++++++\n");
    for (int i = 0; i < const_params.primNum; i++) {
        printf("%d %d %d\n\n",const_params.types[i] ,const_params.bsdfIndexes[i], const_bsdfs[const_params.bsdfIndexes[i]].type);
        
        printf("%lf %lf %lf\n", positions[9 * i], positions[9 * i + 1], positions[9 * i + 2] );
        printf("%lf %lf %lf\n", positions[9 * i + 3], positions[9 * i + 4], positions[9 * i + 5] );
        printf("%lf %lf %lf\n", positions[9 * i + 6], positions[9 * i + 7], positions[9 * i + 8] );
        printf("=======================\n");
        printf("%lf %lf %lf\n", normals[9 * i], normals[9 * i + 1], normals[9 * i + 2] );
        printf("%lf %lf %lf\n", normals[9 * i + 3], normals[9 * i + 4], normals[9 * i + 5] );
        printf("%lf %lf %lf\n", normals[9 * i + 6], normals[9 * i + 7], normals[9 * i + 8] );
        printf("+++++++++++++++++++++++\n\n");
    }
    
}





