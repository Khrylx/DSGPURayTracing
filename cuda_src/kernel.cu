#include "hip/hip_runtime.h"
#include <stdio.h>

#include "helper.cu"
#include "setup.h"

#define MAX_NUM_LIGHT 20
#define MAX_NUM_BSDF 20

__constant__  GPUCamera const_camera;
__constant__  GPUBSDF const_bsdfs[MAX_NUM_BSDF];
__constant__  GPULight const_lights[MAX_NUM_LIGHT];
__constant__  Parameters const_params;



__global__ void
tracePixel()
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index >= const_params.screenW * const_params.screenH) {
        return;
    }
    
    const_params.frameBuffer[3 * index] = 2.0;
    const_params.frameBuffer[3 * index + 1] = 2.0;
    const_params.frameBuffer[3 * index + 2] = 2.0;
}





__global__ void
vectorAdd(float *A, float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        gpuAdd(A + i, B + i, C + i);
        //C[i] = A[i] + B[i];
    }
}

__global__ void
printInfo()
{
    GPUBSDF* bsdfs = const_params.bsdfs;
    GPUCamera* camera = const_params.camera;
    
    for (int i = 0; i < 8; i++) {
        if (bsdfs[i].type == 0) {
            printf("0: %lf %lf %lf\n", bsdfs[i].albedo[0], bsdfs[i].albedo[1], bsdfs[i].albedo[2] );
        }
        else if (bsdfs[i].type == 1) {
            printf("1: %lf %lf %lf\n", bsdfs[i].reflectance[0], bsdfs[i].reflectance[1], bsdfs[i].reflectance[2] );
        }
        else if (bsdfs[i].type == 2) {
            //cout << "2" << endl;
        }
        else if (bsdfs[i].type == 3) {
            printf("3: %lf %lf %lf\n", bsdfs[i].reflectance[0], bsdfs[i].reflectance[1], bsdfs[i].reflectance[2] );
            printf("3: %lf %lf %lf\n", bsdfs[i].transmittance[0], bsdfs[i].transmittance[1], bsdfs[i].transmittance[2] );
        }
        else {
            printf("4: %lf %lf %lf\n", bsdfs[i].albedo[0], bsdfs[i].albedo[1], bsdfs[i].albedo[2] );
        }
    }
    
    
    printf("%lf %lf %lf\n", camera->pos[0], camera->pos[1], camera->pos[2] );
    
    
    float* positions = const_params.positions;
    float* normals = const_params.normals;
    
    printf("+++++++++++++++++++++++\n");
    for (int i = 0; i < const_params.primNum; i++) {
        printf("%d %d %d\n\n",const_params.types[i] ,const_params.bsdfIndexes[i], const_params.bsdfs[const_params.bsdfIndexes[i]].type);
        
        printf("%lf %lf %lf\n", positions[9 * i], positions[9 * i + 1], positions[9 * i + 2] );
        printf("%lf %lf %lf\n", positions[9 * i + 3], positions[9 * i + 4], positions[9 * i + 5] );
        printf("%lf %lf %lf\n", positions[9 * i + 6], positions[9 * i + 7], positions[9 * i + 8] );
        printf("=======================\n");
        printf("%lf %lf %lf\n", normals[9 * i], normals[9 * i + 1], normals[9 * i + 2] );
        printf("%lf %lf %lf\n", normals[9 * i + 3], normals[9 * i + 4], normals[9 * i + 5] );
        printf("%lf %lf %lf\n", normals[9 * i + 6], normals[9 * i + 7], normals[9 * i + 8] );
        printf("+++++++++++++++++++++++\n\n");
    }
    
}





