#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <iostream>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include "helper.cu"
#include "setup.h"
#include <map>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
using namespace std;

__constant__  GPUCamera const_camera;
__constant__  GPUBSDF const_bsdfs[20];


CUDAPathTracer::CUDAPathTracer(PathTracer* _pathTracer)
{
    pathTracer = _pathTracer;
}

CUDAPathTracer::~CUDAPathTracer()
{
    
    hipFree(gpu_types);
    hipFree(gpu_bsdfs);
    hipFree(gpu_positions);
    hipFree(gpu_normals);
}

void CUDAPathTracer::init()
{
    loadCamera();
    loadPrimitives();
}

void CUDAPathTracer::loadCamera()
{
    GPUCamera tmpCam;
    Camera* cam = pathTracer->camera;
    tmpCam.widthDivDist = cam->screenW / cam->screenDist;
    tmpCam.heightDivDist = cam->screenH / cam->screenDist;
    
    for (int i = 0; i < 9; i++) {
        tmpCam.c2w[i] = cam->c2w(i / 3, i % 3);
    }
    
    for (int i = 0; i < 3; i++) {
        tmpCam.pos[i] = cam->pos[i];
    }

    hipError_t err = hipSuccess;
    //hipMalloc((void**)&gpu_camera,sizeof(GPUCamera));
    err = hipMemcpyToSymbol(HIP_SYMBOL(const_camera), &tmpCam,sizeof(GPUCamera));
    
    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void
printInfo()
{
    for (int i = 0; i < 8; i++) {
        if (const_bsdfs[i].type == 0) {
            printf("0: %lf %lf %lf\n", const_bsdfs[i].albedo[0], const_bsdfs[i].albedo[1], const_bsdfs[i].albedo[2] );
        }
        else if (const_bsdfs[i].type == 1) {
            printf("1: %lf %lf %lf\n", const_bsdfs[i].reflectance[0], const_bsdfs[i].reflectance[1], const_bsdfs[i].reflectance[2] );
        }
        else if (const_bsdfs[i].type == 2) {
            //cout << "2" << endl;
        }
        else if (const_bsdfs[i].type == 3) {
            printf("3: %lf %lf %lf\n", const_bsdfs[i].reflectance[0], const_bsdfs[i].reflectance[1], const_bsdfs[i].reflectance[2] );
            printf("3: %lf %lf %lf\n", const_bsdfs[i].transmittance[0], const_bsdfs[i].transmittance[1], const_bsdfs[i].transmittance[2] );
        }
        else {
            printf("4: %lf %lf %lf\n", const_bsdfs[i].albedo[0], const_bsdfs[i].albedo[1], const_bsdfs[i].albedo[2] );
        }
    }

    
    printf("%lf %lf %lf\n", const_camera.pos[0], const_camera.pos[1], const_camera.pos[2] );
    
}


void CUDAPathTracer::loadPrimitives()
{
    vector<Primitive *> primitives;
    for (SceneObject *obj : pathTracer->scene->objects) {
        const vector<Primitive *> &obj_prims = obj->get_primitives();
        primitives.reserve(primitives.size() + obj_prims.size());
        primitives.insert(primitives.end(), obj_prims.begin(), obj_prims.end());
    }
    
    int N = primitives.size();
    int types[N];
    int bsdfs[N];
    float positions[9 * N];
    float normals[9 * N];

    map<BSDF*, int> BSDFMap;
    
    for (int i = 0; i < N; i++) {
        types[i] = primitives[i]->getType();
        BSDF* bsdf  = primitives[i]->get_bsdf();
        
        if (BSDFMap.find(bsdf) == BSDFMap.end()) {
            int index = BSDFMap.size();
            BSDFMap[bsdf] = index;
            bsdfs[i] = index;
        }
        else{
            bsdfs[i] = BSDFMap[bsdf];
        }
        
        
        if (types[i] == 0) {
            Vector3D o = ((Sphere*)primitives[i])->o;
            positions[9 * i] = o[0];
            positions[9 * i + 1] = o[1];
            positions[9 * i + 2] = o[2];
            positions[9 * i + 3] = ((Sphere*)primitives[i])->r;
        }
        else{
            const Mesh* mesh = ((Triangle*)primitives[i])->mesh;
            int v1 = ((Triangle*)primitives[i])->v1;
            int v2 = ((Triangle*)primitives[i])->v2;
            int v3 = ((Triangle*)primitives[i])->v3;
            
            positions[9 * i] = mesh->positions[v1][0];
            positions[9 * i + 1] = mesh->positions[v1][1];
            positions[9 * i + 2] = mesh->positions[v1][2];
            normals[9 * i] = mesh->normals[v1][0];
            normals[9 * i + 1] = mesh->normals[v1][1];
            normals[9 * i + 2] = mesh->normals[v1][2];
            
            positions[9 * i + 3] = mesh->positions[v2][0];
            positions[9 * i + 4] = mesh->positions[v2][1];
            positions[9 * i + 5] = mesh->positions[v2][2];
            normals[9 * i + 3] = mesh->normals[v2][0];
            normals[9 * i + 4] = mesh->normals[v2][1];
            normals[9 * i + 5] = mesh->normals[v2][2];
            
            positions[9 * i + 6] = mesh->positions[v3][0];
            positions[9 * i + 7] = mesh->positions[v3][1];
            positions[9 * i + 8] = mesh->positions[v3][2];
            normals[9 * i + 6] = mesh->normals[v3][0];
            normals[9 * i + 7] = mesh->normals[v3][1];
            normals[9 * i + 8] = mesh->normals[v3][2];
        }
    }
    
    GPUBSDF BSDFArray[BSDFMap.size()];
    
    for (auto itr = BSDFMap.begin(); itr != BSDFMap.end(); itr++) {
        GPUBSDF& gpu_bsdf = BSDFArray[itr->second];
        BSDF* bsdf = itr->first;
        gpu_bsdf.type = bsdf->getType();
        
        if (gpu_bsdf.type == 0) {
            Spectrum& albedo = ((DiffuseBSDF*)bsdf)->albedo;
            gpu_bsdf.albedo[0] = albedo.r;
            gpu_bsdf.albedo[1] = albedo.g;
            gpu_bsdf.albedo[2] = albedo.b;
        }
        else if(gpu_bsdf.type == 1){
            Spectrum& reflectance = ((MirrorBSDF*)bsdf)->reflectance;
            gpu_bsdf.reflectance[0] = reflectance.r;
            gpu_bsdf.reflectance[1] = reflectance.g;
            gpu_bsdf.reflectance[2] = reflectance.b;
        }
        else if(gpu_bsdf.type == 2){
            Spectrum& transmittance = ((RefractionBSDF*)bsdf)->transmittance;
            gpu_bsdf.transmittance[0] = transmittance.r;
            gpu_bsdf.transmittance[1] = transmittance.g;
            gpu_bsdf.transmittance[2] = transmittance.b;
            gpu_bsdf.ior = ((RefractionBSDF*)bsdf)->ior;
        }
        else if(gpu_bsdf.type == 3){
            Spectrum& reflectance = ((GlassBSDF*)bsdf)->reflectance;
            gpu_bsdf.reflectance[0] = reflectance.r;
            gpu_bsdf.reflectance[1] = reflectance.g;
            gpu_bsdf.reflectance[2] = reflectance.b;
            Spectrum& transmittance = ((GlassBSDF*)bsdf)->transmittance;
            gpu_bsdf.transmittance[0] = transmittance.r;
            gpu_bsdf.transmittance[1] = transmittance.g;
            gpu_bsdf.transmittance[2] = transmittance.b;
            gpu_bsdf.ior = ((GlassBSDF*)bsdf)->ior;
        }
        else if(gpu_bsdf.type == 4){
            Spectrum& albedo = ((EmissionBSDF*)bsdf)->radiance;
            gpu_bsdf.albedo[0] = albedo.r;
            gpu_bsdf.albedo[1] = albedo.g;
            gpu_bsdf.albedo[2] = albedo.b;

        }
    }
    
    hipMalloc((void**)&gpu_types, N * sizeof(int));
    hipMalloc((void**)&gpu_bsdfs, N * sizeof(int));
    hipMalloc((void**)&gpu_positions, 9 * N * sizeof(float));
    hipMalloc((void**)&gpu_normals, 9 * N * sizeof(float));
    
    hipMemcpy(gpu_types, types, N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu_bsdfs, bsdfs, N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(gpu_positions, positions, 9 * N * sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(gpu_normals, normals, 9 * N * sizeof(float),hipMemcpyHostToDevice);
    
    //hipMalloc((void**)&gpu_bsdfs, BSDFMap.size() * sizeof(GPUBSDF));
    
        hipError_t err = hipSuccess;
    
    err = hipMemcpyToSymbol(HIP_SYMBOL(const_bsdfs), BSDFArray, BSDFMap.size() * sizeof(GPUBSDF));
    

    
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed! (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    printInfo<<<1, 1>>>();
    hipDeviceSynchronize();
}



extern __global__ void vectorAdd(float *A, float *B, float *C, int numElements);


int test(){

	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED! YEAH!!\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;


}
